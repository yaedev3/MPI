/* File: matmult-cuda-double.cu
 *
 * Purpose: 
 * 
 * Input:
 * 
 * Output:
 * 
 * Compile: nvcc -o matmult-cuda-double.o matmult-cuda-double.cu
 * 
 * Run: ./matmult-cuda-double.o
 * 
 * Algorithm:
 * 
 * Note:
 * 
 * */

 #include <stdio.h>
 #include <hip/hip_runtime.h>
 
 __global__ void VecAdd(double* A, double* B, double* C, int N)
 {
	 int index = blockIdx.x * blockDim.x + threadIdx.x; //indice del vector 
	 int ix; //ix indica el renglon 
	 int iy; //iy toma valores solo entre 0 a N-1
	 double result; //Acumula la suma del renglon por la columna 
	 int k; // Iterador 
	 
	 if(index < N * N)
	 {
		 ix = index / N;
		 iy = index % N;
		 result = 0.0;
 
		 for(k = 0; k < N; k++)
			 result += A[k + N * ix] * B[k * N + iy ];
 
		 C[iy + N * ix] = result;
	 }
 }
 
 // Host code
 int main()
 {
	 //Variables 
	 int N;              // Tamaño de la matriz cuadrada.
	 int i;              // Indice del renglon.
	 int j;              // Indice de la columna.
	 size_t size;        // Tamaño total en memoria.
	 double* h_A;         // Matriz A en el equipo.
	 double* h_B;         // Matriz B en el equipo.
	 double* h_C;         // Matriz C (resultado) en el equipo.
	 double* d_A;         // Matriz A en la memoria de la GPU.
	 double* d_B;         // Matriz B en la memoria de la GPU.
	 double* d_C;         // Matriz C (resultado) en la memoria de la GPU.
	 int Tam;            // Numero de datos que se manejan
	 int NumHilos;       // Hilos por bloque 
	 int numBlock;       // Numero de bloques necesario para procesar los datos 
 
	 //Asignacion de variables
	 N = 5;
	 size = N * sizeof(double) * N;
 
	 //En la memoria del equipo 
	 h_A = (double*)malloc(size);
	 h_B = (double*)malloc(size);
	 h_C = (double*)malloc(size);
	 
	 //En la memoria de la GPU
	 hipMalloc(&d_A, size);
	 hipMalloc(&d_B, size);
	 hipMalloc(&d_C, size);
 
	 //
	 Tam = N * N;
	 NumHilos = 1024;
	 numBlock = Tam / NumHilos; 
 
	 if(Tam % NumHilos > 0) //Si sobran datos, aumenta los bloques en 1
		 numBlock++;
	 
	 // LLena los arreglos A y B
	 for(i = 0;i < N;i++) //Renglon 
		 for(j = 0;j < N;j++) // Columna 
		 {
			 h_A[i + i * j] = rand()%(i + 1);
			 h_B[i + i * j] = rand()%(i + 1);
			 //h_A[j + i * N] = j + i * N + 1;
			 //h_B[j + i * N] = j + i * N + 1;
		 }
 
	 //Copia los arreglos de memoria del CPU a memoria de la GPU 
	 hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	 hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
 
	 // Invoke kernel
	 VecAdd<<<numBlock, NumHilos >>>(d_A, d_B, d_C, N);
 
	 
	 //Copea el resultado de la multiplicacion de memoria de la GPU a memoria de la CPU
	 hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	 
	 //Imprime la matriz A
	 printf("Matriz A\n");
	 for(i = 0;i < N;i++)
	 {
		 for(j = 0;j < N;j++)
			 printf("%.2e ", h_A[j + i * N]);
		 printf("\n");
	 }
	 
	 //Imprime la matriz B
	 printf("Matriz B\n");
	 for(i = 0;i < N;i++)
	 {
		 for(j = 0;j < N;j++)
			 printf("%.2e ", h_B[j + i * N]);
		 printf("\n");
	 }
	 
	 //Imprime la matriz C
	 printf("Matriz C\n");
	 for(i = 0;i < N;i++)
	 {
		 for(j = 0;j < N;j++)
			 printf("%.2e ", h_C[j + i * N]);
		 printf("\n");
	 }
 
	 //Libera la memoria utilizada.
	 // Free device memory
	 hipFree(d_A);
	 hipFree(d_B);
	 hipFree(d_C);
	 // Free host memory
	 free(h_A);
	 free(h_B);
	 free(h_C);
 }
